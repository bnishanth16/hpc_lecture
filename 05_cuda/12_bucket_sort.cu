// Name: Nishanth Baskaran
// Student ID: 19M15017
// HPSC Assignment-L5

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void init(int *bucket) {
  int i= blockIdx.x * blockDim.x + threadIdx.x;
  bucket[i]=0;
}

__global__ void add(int *key,int *bucket){
  int i= blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]],1);
}

__global__ void sort(int *key,int *bucket){
  int i= blockIdx.x * blockDim.x + threadIdx.x;

  for (int j=0,k=0; k<=i; j++){
    key[i]=j;
    __syncthreads();
    k+=bucket[j];
    __syncthreads();
  } 
}

int main() {
  int n = 50;
  int range = 5;

  int *key, *bucket;
  hipMallocManaged(&key,n*sizeof(int));
  hipMallocManaged(&bucket,range*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  //since the range and n is small, using only 1 block for parallelisation
  init<<<1,range>>>(bucket);
  add<<<1,n>>>(key,bucket);
  sort<<<1,n>>>(key,bucket);
  hipDeviceSynchronize();
  
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(key);
  hipFree(bucket);
}
