// HPSC Final Report
// Name: Nishanth Baskaran
// Student id: 19M15017
#include <iostream>
#include <vector>
#include <chrono>
#include "solver.h"

using namespace std;

int main()
{
  const int nx = 41;  const int ny = 41;
  const int nt = 700; const int nit = 50; 
  
  const double L = 2.0;
  const double dx = L/(nx-1);    
  const double dy = L/(ny-1);
  
  const double rho = 1.0;
  const double nu = 0.1;
  const double dt = 0.001;
    
  int size = nx*ny*sizeof(double);
  double *u, *un, *v, *vn, *p, *pn, *b;

  hipMallocManaged(&u,size);   hipMallocManaged(&un,size);
  hipMallocManaged(&v,size);   hipMallocManaged(&vn,size);
  hipMallocManaged(&p,size);   hipMallocManaged(&pn,size);
  hipMallocManaged(&b,size);

  initialize(u,un,v,vn,p,pn,b,nx,ny);

  dim3 threadsPerBlock(128,1);
  dim3 blockNumber ((nx+threadsPerBlock.x - 1)/threadsPerBlock.x, (ny+threadsPerBlock.y - 1)/threadsPerBlock.y);

  auto t_initial = chrono::steady_clock::now();

  for (int iter = 0; iter < nt; iter++)
  {
    build_up_b<<<blockNumber,threadsPerBlock>>>(b,u,v,rho,dt,dx,dy,nx,ny);
    hipDeviceSynchronize();

    for (int p_iter = 0; p_iter < nit; p_iter++)
    {
      pressure_poisson<<<blockNumber,threadsPerBlock>>>(p,pn,b,rho,dt,dx,dy,nx,ny);
      boundary_pressure<<<blockNumber,threadsPerBlock>>>(p,nx,ny);
      copy_function(pn,p,nx,ny);
      hipDeviceSynchronize();
    }
    
    velocity_solver<<<blockNumber,threadsPerBlock>>>(u,un,v,vn,p,pn,b,rho,nu,dt,dx,dy,nx,ny);
    boundary_velocity<<<blockNumber,threadsPerBlock>>>(u,v,nx,ny);
    copy_function(un,u,nx,ny);    copy_function(vn,v,nx,ny);
    hipDeviceSynchronize();
  }
  auto t_final = chrono::steady_clock::now();
  double time = chrono::duration<double>(t_final-t_initial).count();
  cout << "time = " << time << endl;
  save_result(u,v,p,nx,ny);

  hipFree(u); hipFree(un); hipFree(v); hipFree(vn); hipFree(p); hipFree(pn); hipFree(b);
  return 0;
}

